#include "hip/hip_runtime.h"
//=======================================================================
// Copyright Ahmed Magdi Osman 2015.
// Distributed under the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

/*
 * Image3.cu
 *
 *  Created on: Oct 16, 2014
 *      Author: Ahmed M. Osman
 */

#include "Image3.hpp"

CUDA_CALLABLE_MEMBER Image3::Image3(int width, int height) {
	imWidth = width;
	imHeight = height;
	imSize = imWidth * imHeight;
	data = new uchar3[imSize];
}

Image3::Image3(const CImg<unsigned char>& _img, bool border) {
	//Creates an Image3 instance of a CImg datatype
	//border creates a border with 1 pixel width
	// TODO: make the border thickness a parameter
	if (border) {
		imWidth = _img.width() + 2 * BORDERTHICKNESS;
		imHeight = _img.height() + 2 * BORDERTHICKNESS;
		imSize = imWidth * imHeight;
		data = new uchar3[imSize](); //() initializes elements to zero

		//Copy elements
		int j = imWidth + BORDERTHICKNESS, x = 0, y = 0;
		for (; y < _img.height(); y++) {

			for (x = 0; x < _img.width(); x++, j++) {
				data[j].x = _img(x, y, 0, 0);
				data[j].y = _img(x, y, 0, 1);
				data[j].z = _img(x, y, 0, 2);

			}
			j += 2 * BORDERTHICKNESS;	//Shift 2 border pixels
			//	j = (y + 2) *  imWidth * NUMCHANNELS + NUMCHANNELS;	// Same thing but  just more complicated
		}
	} else {
		imWidth = _img.width();
		imHeight = _img.height();
		imSize = imWidth * imHeight;
		data = new uchar3[imSize];

		//Copy elements
		unsigned int i = 0, x = 0, y = 0;
		for (; i < imSize; i++) {
			x = i % imWidth;
			y = i / imWidth;
			data[i].x = _img(x, y, 0, 0);
			data[i].y = _img(x, y, 0, 1);
			data[i].z = _img(x, y, 0, 2);
		}
	}
}

CUDA_CALLABLE_MEMBER Image3::Image3() {
	data = 0;
	imHeight = 0;
	imWidth = 0;
	imSize = 0;

}

CUDA_CALLABLE_MEMBER Image3::Image3(const Image3& _img)

{

	imWidth = _img.width();
	imHeight = _img.height();
	imSize = imWidth * imHeight;
	data = new uchar3[imSize];

	//Copy elements
	const uchar3* _imgPtr = _img.getConstPtr();
	unsigned int i = 0;
	for (i = 0; i < imSize; i++) {
		data[i].x = _imgPtr[i].x;
		data[i].y = _imgPtr[i].y;
		data[i].z = _imgPtr[i].z;
	}
}

CUDA_CALLABLE_MEMBER Image3::~Image3() {
	delete[] data;
	data = 0;	//Set pointer to NULL
}
