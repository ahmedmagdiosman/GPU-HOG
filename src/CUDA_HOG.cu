#include "hip/hip_runtime.h"
//=======================================================================
// Copyright Ahmed Magdi Osman 2015.
// Distributed under the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

/*
 * CUDA_HOG.cu
 *
 *  Created on: Oct 27, 2014
 *      Author: Ahmed M. Osman
 */
#include "CUDA_HOG.hpp"

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
		hipError_t _m_cudaStat = value;										\
		if (_m_cudaStat != hipSuccess) {										\
			fprintf(stderr, "Error %s at line %d in file %s\n",					\
					hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
					exit(1);															\
		} }

__global__ void CUDA::computeGradient(const Image3* _img, Vec2* _imgGradient) {
	//TODO:  handle leftovers. implement shared mem
	if (threadIdx.y < blockDim.y && threadIdx.x < blockDim.x) {

		uint2 blkShr;	// Each block's share //-2 to remove border padding
		blkShr.x = (_img->width() - 2 * BORDERTHICKNESS) / gridDim.x;
		blkShr.y = (_img->height() - 2 * BORDERTHICKNESS) / gridDim.y;
		uint2 tdShr; //Each thread's share;
		tdShr.x = blkShr.x / blockDim.x;
		tdShr.y = blkShr.y / blockDim.y;
		uint2 idx;
		idx.x = blockIdx.x * blkShr.x + threadIdx.x * tdShr.x;
		idx.y = blockIdx.y * blkShr.y + threadIdx.y * tdShr.y;

		//		 Gradient computation -1 0 1 in x and y
		float2* imgGradientPtr;
		uchar3 *prevImgPtr = _img->getConstPtr(1, idx.y), *imgPtr =
				_img->getConstPtr(1, idx.y + 1), *nextImgPtr;
		float gxR, gxG, gxB, gyR, gyG, gyB, mag, mR, mG, mB;
		int x = idx.x, y = idx.y;

		for (; y < idx.y + tdShr.y; y++) {

			imgGradientPtr = _imgGradient->getPtr(0, y);		//row pointer
			nextImgPtr = _img->getConstPtr(1, y + 2); // +1 for border. +1 for nextImgPtr.
			for (x = idx.x; x < idx.x + tdShr.x; x++) {

				//			//x gradient
				gxR = sqrtf(imgPtr[x + 1].x) - sqrtf(imgPtr[x - 1].x);
				gxG = sqrtf(imgPtr[x + 1].y) - sqrtf(imgPtr[x - 1].y);
				gxB = sqrtf(imgPtr[x + 1].z) - sqrtf(imgPtr[x - 1].z);

				//			//y gradient
				gyR = sqrtf(nextImgPtr[x].x) - sqrtf(prevImgPtr[x].x);
				gyG = sqrtf(nextImgPtr[x].y) - sqrtf(prevImgPtr[x].y);
				gyB = sqrtf(nextImgPtr[x].z) - sqrtf(prevImgPtr[x].z);

				//magnitude
				mR = sqrtf(gxR * gxR + gyR * gyR);
				mG = sqrtf(gxG * gxG + gyG * gyG);
				mB = sqrtf(gxB * gxB + gyB * gyB);

				mag = max(max(mR, mG), mB);
				//EXPERIMENTAL: REMOVED ABS in X
				if (mag == mR) {
					imgGradientPtr[x].x = mR;
					imgGradientPtr[x].y = atan2f(fabsf(gyR), ((gxR) + EPSILON))
									* 180 / PI; //atan(y/x) returns value in radians. NOTE: atan returns values from -pi/2 to pi/2, atan2 returns values from -pi to pi. fabsf gives absolute value. Absolute value is used according to HOG paper for unsigned orientations
				} else if (mag == mG) {
					imgGradientPtr[x].x = mG;
					imgGradientPtr[x].y = atan2f(fabsf(gyG), ((gxG) + EPSILON))
									* 180 / PI;
				} else	//mag == mB
				{
					imgGradientPtr[x].x = mB;
					imgGradientPtr[x].y = atan2f(fabsf(gyB), ((gxB) + EPSILON))
									* 180 / PI;
				}

			}
			prevImgPtr = imgPtr;
			imgPtr = nextImgPtr;

		}

		return;
	}
}
__device__ inline void CUDA::resetHistogram(HOGDescriptor& blockHist) {
	blockHist.bin[0] = 0;
	blockHist.bin[1] = 0;
	blockHist.bin[2] = 0;
	blockHist.bin[3] = 0;
	blockHist.bin[4] = 0;
	blockHist.bin[5] = 0;
	blockHist.bin[6] = 0;
	blockHist.bin[7] = 0;
	blockHist.bin[8] = 0;
	blockHist.bin[9] = 0;
	blockHist.bin[10] = 0;
	blockHist.bin[11] = 0;
	blockHist.bin[12] = 0;
	blockHist.bin[13] = 0;
	blockHist.bin[14] = 0;
	blockHist.bin[15] = 0;
	blockHist.bin[16] = 0;
	blockHist.bin[17] = 0;
	blockHist.bin[18] = 0;
	blockHist.bin[19] = 0;
	blockHist.bin[20] = 0;
	blockHist.bin[21] = 0;
	blockHist.bin[22] = 0;
	blockHist.bin[23] = 0;
	blockHist.bin[24] = 0;
	blockHist.bin[25] = 0;
	blockHist.bin[26] = 0;
	blockHist.bin[27] = 0;
	blockHist.bin[28] = 0;
	blockHist.bin[29] = 0;
	blockHist.bin[30] = 0;
	blockHist.bin[31] = 0;
	blockHist.bin[32] = 0;
	blockHist.bin[33] = 0;
	blockHist.bin[34] = 0;
	blockHist.bin[35] = 0;

}
__global__ void CUDA::computeHistogram(const Vec2* _imgGradient,
		HOGDescriptor* g_hogDsc, const uint2 _numWindows) {
	// 		Orientation binning: magnitude added to orientation bin. Orientation = atan(y/x) INTERPOLATION
	extern __shared__ HOGDescriptor s_blockHist[];
	unsigned int s_idx = threadIdx.y * blockDim.x + threadIdx.x;//index in shared block histogram array
	resetHistogram(s_blockHist[s_idx]); //explicitly sets bins to zero

	uint2 wIdx = make_uint2((blockIdx.z % _numWindows.x) * WINDOWSTRIDEX,
			(blockIdx.z / _numWindows.x) * WINDOWSTRIDEY); //Window index
	uint2 bIdx = make_uint2(blockIdx.x * BLOCKSTRIDEX,
			blockIdx.y * BLOCKSTRIDEY); // block index in a window
	uint2 pIdx = make_uint2(wIdx.x + bIdx.x + threadIdx.x,
			wIdx.y + bIdx.y + threadIdx.y); //pixel index (location) in the image;

	uint2 tdShr = make_uint2(1, 1); // for now only 1 pixel is processed per thread.
	float mag, angle, a1, a2, b1, b2, c1, c2;

	const uint2 halfCell = make_uint2(CELLSIZEX / 2, CELLSIZEY / 2);

	uint2 cellLowCen = make_uint2(wIdx.x + bIdx.x + halfCell.x,
			wIdx.y + bIdx.y + halfCell.y), cellHighCen = make_uint2(
					wIdx.x + bIdx.x + halfCell.x + CELLSIZEX,
					wIdx.y + bIdx.y + halfCell.y + CELLSIZEY);
	int lowBin, z1, z2, xy11, xy12, xy21, xy22;

	int bx = WINDOWSIZEX / NUMCELLSX, by = WINDOWSIZEY / NUMCELLSY, bz =
			ANGLERANGE / NBINS;
	const int halfBin = bz / 2.f;
	float2* imgGradientPtr;

	unsigned int y, x;
	for (y = 0; y < tdShr.y; y++) {
		imgGradientPtr = _imgGradient->getConstPtr(pIdx.x, pIdx.y + y);
		for (x = 0; x < tdShr.x; x++) {

			mag = imgGradientPtr[x].x;
			angle = imgGradientPtr[x].y;

			z1 = (angle) / (bz + EPSILON);
			lowBin = z1 * bz + halfBin;
			if (lowBin > angle) {
				lowBin -= bz;
			}
			//precompute weights
			uint2 pos = make_uint2(x + pIdx.x, y + pIdx.y);
			a1 = pos.x > cellLowCen.x ? (float) (pos.x - cellLowCen.x) / bx : 0;
			b1 = (pos.y) > cellLowCen.y ?
					(float) (pos.y - cellLowCen.y) / by : 0;
			c1 = (angle - lowBin) / bz;
			a2 = (pos.x) < cellHighCen.x ? 1 - a1 : 0;
			b2 = (pos.y) < cellHighCen.y ? 1 - b1 : 0;
			c2 = 1 - c1;

			xy11 = 0;
			xy21 = NBINS * 1;
			xy12 = NBINS * 2;
			xy22 = NBINS * 3;

			lowBin = (lowBin % ANGLERANGE + ANGLERANGE) % ANGLERANGE; // to handle negative modulo e.g. -10%180 = 170
			z1 = lowBin / bz;
			z2 = (z1 + 1) % NBINS;

			// storing trilinear interpolation and binning
			s_blockHist[s_idx].bin[xy11 + z1] += mag * a2 * b2 * c2;
			s_blockHist[s_idx].bin[xy11 + z2] += mag * a2 * b2 * c1;
			s_blockHist[s_idx].bin[xy12 + z1] += mag * a2 * b1 * c2;
			s_blockHist[s_idx].bin[xy21 + z1] += mag * a1 * b2 * c2;
			s_blockHist[s_idx].bin[xy12 + z2] += mag * a2 * b1 * c1;
			s_blockHist[s_idx].bin[xy21 + z2] += mag * a1 * b2 * c1;
			s_blockHist[s_idx].bin[xy22 + z1] += mag * a1 * b1 * c2;
			s_blockHist[s_idx].bin[xy22 + z2] += mag * a1 * b1 * c1;

		}
	}
	__syncthreads();

	//////////////////////////////////////////////////////////
	//Reduction												//
	//////////////////////////////////////////////////////////
	const unsigned int thblockSize = blockDim.x * blockDim.y;
	const unsigned int HBLOCKSIZE = NBINS * BLOCKSIZEX * BLOCKSIZEY;
	unsigned int index;
	for (unsigned int s = 1; s < thblockSize; s *= 2) {
		index = s_idx * s * 2;
		if (index < thblockSize) {
#pragma unroll 36 //static unrolling of block descriptors
			for (unsigned int i = 0; i < HBLOCKSIZE; i++)
				s_blockHist[index].bin[i] += s_blockHist[index + s].bin[i];

		}
		__syncthreads();
	}
	////////////////////////////////////////////////////////////
	//Copying final reduced block descriptor to global memory //
	////////////////////////////////////////////////////////////

	// changing indices from image coordinations to numbers. e.g. window starting at (32,128) becomes index (1,2)
	wIdx = make_uint2(wIdx.x / WINDOWSTRIDEX, wIdx.y / WINDOWSTRIDEY);
	// changing indices from image coordinations to numbers. e.g. block starting at (8,8) becomes index (1,1)
	bIdx = make_uint2(bIdx.x / BLOCKSTRIDEX, bIdx.y / BLOCKSTRIDEY);

	const unsigned int NUMBLOCKS = (NUMBLOCKSX * NUMBLOCKSY);

	index = blockIdx.z * NUMBLOCKS + (blockIdx.y * gridDim.x + blockIdx.x);
	if (s_idx < HBLOCKSIZE) {
		g_hogDsc[index].bin[s_idx] = s_blockHist[0].bin[s_idx];

	}

	return;
}
__global__ void CUDA::normalizeHistogram(HOGDescriptor* g_hogDsc) {
	////////////////////////////////////////////////////////////
	//BLOCK NORMALIZATION									  //
	////////////////////////////////////////////////////////////

	// copying histograms from global to shared memory
	__shared__ HOGDescriptor s_blockHist;
	__shared__ HOGDescriptor s_tempBlockHist;
	const unsigned int NUMBLOCKS = (NUMBLOCKSX * NUMBLOCKSY);
	unsigned int idx = blockIdx.z * NUMBLOCKS
			+ (blockIdx.y * gridDim.x + blockIdx.x);
	s_blockHist.bin[threadIdx.x] = g_hogDsc[idx].bin[threadIdx.x];
	s_tempBlockHist.bin[threadIdx.x] = s_blockHist.bin[threadIdx.x]
	                                                   * s_blockHist.bin[threadIdx.x]; //squaring each element
	__syncthreads();
	unsigned int index;

	//Calculating sum of squares using parallel reduction
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		index = threadIdx.x * s * 2;
		if (index < blockDim.x) {
			s_tempBlockHist.bin[index] = s_tempBlockHist.bin[index]
			                                                 + s_tempBlockHist.bin[index + s];

		}
		__syncthreads();
	}

	float sumOfSqr = s_tempBlockHist.bin[0]; // saving sum of sqr in a temp variable. Must sync threads before bin[0] is modified
	__syncthreads();
	//Normalize each element and cap to 0.2

	s_tempBlockHist.bin[threadIdx.x] =
			min(
					(s_blockHist.bin[threadIdx.x]
					                 / sqrtf(sumOfSqr + EPSILON * EPSILON)),
					                 L2THRESHOLD);

	//copying first-normalized values
	s_blockHist.bin[threadIdx.x] = s_tempBlockHist.bin[threadIdx.x];
	//Squaring each element
	s_tempBlockHist.bin[threadIdx.x] *= s_tempBlockHist.bin[threadIdx.x];
	__syncthreads();

	//Calculating sum of squares using parallel reduction, again ..
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		index = threadIdx.x * s * 2;
		if (index < blockDim.x) {
			s_tempBlockHist.bin[index] = s_tempBlockHist.bin[index]
			                                                 + s_tempBlockHist.bin[index + s];

		}
		__syncthreads();
	}

	sumOfSqr = s_tempBlockHist.bin[0]; // saving sum of sqr in a temp variable. Must sync threads before bin[0] is modified
	__syncthreads();
	//Normalize each element
	s_blockHist.bin[threadIdx.x] = (s_blockHist.bin[threadIdx.x]
	                                                / sqrtf(sumOfSqr + EPSILON * EPSILON));

	//Copy to global memory
	g_hogDsc[idx].bin[threadIdx.x] = s_blockHist.bin[threadIdx.x];

}

CUDA_CALLABLE_MEMBER CUDA_HOG::CUDA_HOG(const Image3& _img) :
				img(_img), imgGradient(img.width() - 2 * PADTHICKNESS,
						img.height() - 2 * PADTHICKNESS)

{

	numWindowsX = imgGradient.width() / WINDOWSTRIDEX - 1;
	numWindowsY = imgGradient.height() / WINDOWSTRIDEY - 1;
	hogDsc = new HOGDescriptor[numWindowsX * numWindowsY * NUMBLOCKSX
	                           * NUMBLOCKSY]; // total number of blocks. By default (7x15 blocks per window) * (number of windows in the image)

}

void CUDA_HOG::writeFeatureVector(unsigned int _windowX,
		unsigned int _windowY) {
	struct stat st;
	if (stat("GPU_features", &st) == -1) {
		mkdir("GPU_features", 0777);
	}
	std::ofstream outfile;
	std::stringstream ss1, ss2;
	ss1 << _windowX;
	ss2 << _windowY;
	unsigned int windowIdx = (_windowY * numWindowsX + _windowX) * NUMBLOCKSX
			* NUMBLOCKSY;
	std::string name = "GPU_features/vector_" + ss1.str() + "_" + ss2.str()
					+ ".txt";
	outfile.open(name.c_str());
	for (unsigned int i = 0; i < (NUMBLOCKSX * NUMBLOCKSY); i++) {
#pragma unroll 36
		for (unsigned int j = 0; j < 36; j++) {
			outfile << hogDsc[windowIdx + i].bin[j] << "\n";
		}

		//outfile << "\n==========BLOCK END ==========\n";
	}
	outfile.close();
}

void CUDA_HOG::writeGradient() {

	std::ofstream outfile;

	std::string name = "gradient.txt";
	outfile.open(name.c_str());
	float2* grdPtr;
	for (unsigned int y = 0; y < imgGradient.height(); y++) {
		grdPtr = imgGradient.getConstPtr(0, y);
		for (unsigned int x = 0; x < imgGradient.width(); x++) {
			outfile << "M: " << grdPtr[x].x << "A: " << grdPtr[x].y << "\t";
		}
		outfile << "\n";
	}
	outfile.close();
}
void CUDA_HOG::computeHOG() {
	//TODO: change kernel configs to match input. Currently works with 640x480 only
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	Image3* d_img;

	// allocate and copy image object to device ==> shallow copy
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_img, sizeof(Image3)));
	CUDA_CHECK_RETURN(
			hipMemcpy(d_img, &(img), sizeof(Image3), hipMemcpyHostToDevice));
	uchar3* d_imgPtr, *h_imgPtr = img.getConstPtr();

	//allocate and copy image data to device image-data pointer d_imgPtr ==>deep copy
	CUDA_CHECK_RETURN(
			hipMalloc((void** ) &(d_imgPtr), sizeof(uchar3) * img.size()));
	CUDA_CHECK_RETURN(
			hipMemcpy(d_imgPtr, h_imgPtr, sizeof(uchar3) * img.size(),
					hipMemcpyHostToDevice)); //element  copy

	//copy device image-data pointer to device image member pointer
	CUDA_CHECK_RETURN(
			hipMemcpy(&(d_img->data), &d_imgPtr, sizeof(uchar3*),
					hipMemcpyHostToDevice));

	//////////////////////////////////////////////////////////////////////////////////////
	//Gradient calculation																//
	//////////////////////////////////////////////////////////////////////////////////////
	Vec2* d_imgGradient;
	// allocate and copy gradient vector object to device ==> shallow copy
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_imgGradient, sizeof(Vec2)));
	CUDA_CHECK_RETURN(
			hipMemcpy(d_imgGradient, &(imgGradient), sizeof(Vec2),
					hipMemcpyHostToDevice));

	float2* d_imgGradientPtr;
	//allocating memory for gradient vector
	CUDA_CHECK_RETURN(
			hipMalloc((void** ) &(d_imgGradientPtr),
					sizeof(float2) * imgGradient.size()));

	//copy gradient pointer to the member pointer of the gradient object
	CUDA_CHECK_RETURN(
			hipMemcpy(&(d_imgGradient->data), &d_imgGradientPtr,
					sizeof(float2*), hipMemcpyHostToDevice));
	dim3 gridSize = dim3(20, 20, 1);
	dim3 blockSize = dim3(32, 24, 1);

	hipEventRecord(start);
	CUDA::computeGradient<<<gridSize, blockSize>>>(d_img, d_imgGradient);
	hipEventRecord(stop);
	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Time for gradient kernel: %f ms\n", milliseconds);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	CUDA_CHECK_RETURN(
			hipMemcpy((imgGradient.data), (d_imgGradientPtr),
					sizeof(float2) * imgGradient.size(),
					hipMemcpyDeviceToHost))
	CUDA_CHECK_RETURN(hipGetLastError());

	//////////////////////////////////////////////////////////////////////////////////////
	//Histogram computation																//
	//////////////////////////////////////////////////////////////////////////////////////
	HOGDescriptor* d_hogDsc;
	//Allocating memory for HOG descriptor
	size_t totalBlockNum = numWindowsX * numWindowsY * NUMBLOCKSX * NUMBLOCKSY;
	CUDA_CHECK_RETURN(
			hipMalloc((void** )&d_hogDsc,
					sizeof(HOGDescriptor) * totalBlockNum));

	//explicit initialization by zero
	CUDA_CHECK_RETURN(
			hipMemset(d_hogDsc, 0, sizeof(HOGDescriptor) * totalBlockNum));

	gridSize = dim3(NUMBLOCKSX, NUMBLOCKSY, numWindowsX * numWindowsY);	//hog.numWindowsX*hog.numWindowsY thread block per each hog block. z dimension has number of windows per image
	blockSize = dim3(CELLSIZEX * BLOCKSIZEX, CELLSIZEY * BLOCKSIZEY, 1);// thread per pixel
	uint2 numWindows = make_uint2(numWindowsX, numWindowsY);
	hipEventRecord(start);
	CUDA::computeHistogram<<<gridSize,blockSize,sizeof(HOGDescriptor)*blockSize.x*blockSize.y>>>(d_imgGradient, d_hogDsc, numWindows );
	hipEventRecord(stop);
	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Time for histogram kernel: %f ms\n", milliseconds);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());

	CUDA_CHECK_RETURN(
			hipMemcpy(hogDsc, d_hogDsc, sizeof(HOGDescriptor) * totalBlockNum,
					hipMemcpyDeviceToHost));

	//////////////////////////////////////////////////////////////////////////////////////
	//Histogram normalization															//
	//////////////////////////////////////////////////////////////////////////////////////
	blockSize = dim3(NBINS * BLOCKSIZEX * BLOCKSIZEY, 1, 1); // thread per histogram bin. by default 36 threads
	hipEventRecord(start);
	CUDA::normalizeHistogram<<<gridSize, blockSize>>>(d_hogDsc);
	hipEventRecord(stop);
	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Time for normalization kernel: %f ms\n", milliseconds);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());

	CUDA_CHECK_RETURN(
			hipMemcpy(hogDsc, d_hogDsc, sizeof(HOGDescriptor) * totalBlockNum,
					hipMemcpyDeviceToHost));

	unsigned int x, y;
	for (y = 0; y < numWindows.y; y++) {
		for (x = 0; x < numWindows.x; x++) {
			writeFeatureVector(x, y);

		}
	}
	writeGradient();
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_img);
	hipFree(d_imgPtr);
	hipFree(d_imgGradient);
	hipFree(d_imgGradientPtr);
	hipFree(d_hogDsc);

}

CUDA_HOG::~CUDA_HOG() {

	delete[] hogDsc;
	hogDsc = 0;
}
