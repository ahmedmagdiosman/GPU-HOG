#include "hip/hip_runtime.h"
//=======================================================================
// Copyright Ahmed Magdi Osman 2015.
// Distributed under the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

/*
 * Vec2.cu
 *
 *  Created on: Oct 16, 2014
 *      Author: Ahmed M. Osman
 */
#include "Vec2.hpp"

CUDA_CALLABLE_MEMBER Vec2::Vec2(int width, int height) {
	imWidth = width;
	imHeight = height;
	imSize = imWidth * imHeight;
	data = new float2[imSize];
}

CUDA_CALLABLE_MEMBER Vec2::Vec2() {
	data = 0;
	imHeight = 0;
	imWidth = 0;
	imSize = 0;

}

CUDA_CALLABLE_MEMBER Vec2::Vec2(const Vec2& _vec)

{

	imWidth = _vec.width();
	imHeight = _vec.height();
	imSize = imWidth * imHeight;
	data = new float2[imSize];

	//Copy elements
	const float2* _vecPtr = _vec.getConstPtr();
	unsigned int i = 0;
	for (i = 0; i < imSize; i++) {
		data[i].x = _vecPtr[i].x;
		data[i].y = _vecPtr[i].y;

	}
}

CUDA_CALLABLE_MEMBER Vec2::~Vec2() {
	delete[] data;
	data = 0;	//Set pointer to NULL
}
