//=======================================================================
// Copyright Ahmed Magdi Osman 2015.
// Distributed under the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

/*
 * main.cu
 *
 *  Created on: Oct 27, 2014
 *      Author: Ahmed M. Osman
 */

#include <stdio.h>
#include <stdlib.h>
#include "CImg.h"
#include "Image3.hpp"
#include "Vec2.hpp"
#include  "CUDA_HOG.hpp"
using namespace cimg_library;

int main(int argc, char* argv[]) {

	if (argc < 2) {
		std::cerr << "Usage: " << argv[0] << "[IMAGE NAME]" << std::endl;

		return 1;
	}

	//Reads image
	CImg<unsigned char> srcImg(argv[1]);

	//Convert from CImg class to a simple handmade class.
	Image3 img = Image3(srcImg, true); // 'true' pads the image
	//Initialize HOG instance
	CUDA_HOG hog = CUDA_HOG(img);
	//Compute a sliding window HOG on the default image scale with default parameters of [Dalal 2005]
	hog.computeHOG();

	//	//TODO: FIX FOR SMALLER IMAGES. CURRENTLY FAILS KERNEL LAUNCH
	//	srcImg.resize_halfXY();
	//	Image3 halfImg = Image3(srcImg, true);
	//	CUDA_HOG halfHog = CUDA_HOG(halfImg);
	//	halfHog.computeHOG();
	//
	//	srcImg.resize_halfXY();
	//	Image3 qrtImg = Image3(srcImg, true);
	//	CUDA_HOG qrtHog = CUDA_HOG(qrtImg);
	//	qrtHog.computeHOG();

	hipDeviceReset();

	return 0;
}
